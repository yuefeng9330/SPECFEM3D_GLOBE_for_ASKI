
#include <hip/hip_runtime.h>
// from compute_stacey_acoustic_cuda.cu
#define NGLLX 5
#define NGLL2 25
#define INDEX2(xsize,x,y) x + (y)*xsize
#define INDEX4(xsize,ysize,zsize,x,y,z,i) x + xsize*(y + ysize*(z + zsize*i))

typedef float realw;

__global__ void compute_stacey_acoustic_kernel(realw* potential_dot_acoustic,
                                               realw* potential_dot_dot_acoustic,
                                               int interface_type,
                                               int num_abs_boundary_faces,
                                               int* abs_boundary_ispec,
                                               int* nkmin_xi,
                                               int* nkmin_eta,
                                               int* njmin,
                                               int* njmax,
                                               int* nimin,
                                               int* nimax,
                                               realw* abs_boundary_jacobian2D,
                                               realw* wgllwgll,
                                               int* ibool,
                                               realw* vpstore,
                                               int SAVE_FORWARD,
                                               realw* b_absorb_potential) {

  int igll = threadIdx.x;
  int iface = blockIdx.x + gridDim.x*blockIdx.y;

  int i,j,k,iglob,ispec;
  realw sn;
  realw jacobianw,fac1;

  // don't compute points outside NGLLSQUARE==NGLL2==25
  // way 2: no further check needed since blocksize = 25
  if (iface < num_abs_boundary_faces){

  //  if(igll<NGLL2 && iface < num_abs_boundary_faces) {

    // "-1" from index values to convert from Fortran-> C indexing
    ispec = abs_boundary_ispec[iface]-1;

    // determines indices i,j,k depending on absorbing boundary type
    switch( interface_type ){
      case 4:
        // xmin
        if (nkmin_xi[INDEX2(2,0,iface)] == 0 || njmin[INDEX2(2,0,iface)] == 0) return;

        i = 0; // index -1
        k = (igll/NGLLX);
        j = (igll-k*NGLLX);

        if (k < nkmin_xi[INDEX2(2,0,iface)]-1 || k > NGLLX-1) return;
        if (j < njmin[INDEX2(2,0,iface)]-1 || j > njmax[INDEX2(2,0,iface)]-1) return;

        fac1 = wgllwgll[k*NGLLX+j];
        break;

      case 5:
        // xmax
        if (nkmin_xi[INDEX2(2,1,iface)] == 0 || njmin[INDEX2(2,1,iface)] == 0) return;

        i = NGLLX-1;
        k = (igll/NGLLX);
        j = (igll-k*NGLLX);

        if (k < nkmin_xi[INDEX2(2,1,iface)]-1 || k > NGLLX-1) return;
        if (j < njmin[INDEX2(2,1,iface)]-1 || j > njmax[INDEX2(2,1,iface)]-1) return;

        fac1 = wgllwgll[k*NGLLX+j];
        break;

      case 6:
        // ymin
        if (nkmin_eta[INDEX2(2,0,iface)] == 0 || nimin[INDEX2(2,0,iface)] == 0) return;

        j = 0;
        k = (igll/NGLLX);
        i = (igll-k*NGLLX);

        if (k < nkmin_eta[INDEX2(2,0,iface)]-1 || k > NGLLX-1) return;
        if (i < nimin[INDEX2(2,0,iface)]-1 || i > nimax[INDEX2(2,0,iface)]-1) return;

        fac1 = wgllwgll[k*NGLLX+i];
        break;

      case 7:
        // ymax
        if (nkmin_eta[INDEX2(2,1,iface)] == 0 || nimin[INDEX2(2,1,iface)] == 0) return;

        j = NGLLX-1;
        k = (igll/NGLLX);
        i = (igll-k*NGLLX);

        if (k < nkmin_eta[INDEX2(2,1,iface)]-1 || k > NGLLX-1) return;
        if (i < nimin[INDEX2(2,1,iface)]-1 || i > nimax[INDEX2(2,1,iface)]-1) return;

        fac1 = wgllwgll[k*NGLLX+i];
        break;

      case 8:
        // zmin
        k = 0;
        j = (igll/NGLLX);
        i = (igll-j*NGLLX);

        if (j < 0 || j > NGLLX-1) return;
        if (i < 0 || i > NGLLX-1) return;

        fac1 = wgllwgll[j*NGLLX+i];
        break;

    }

    iglob = ibool[INDEX4(5,5,5,i,j,k,ispec)]-1;

    // determines bulk sound speed
    // velocity
    sn = potential_dot_acoustic[iglob] / vpstore[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)] ;

    // gets associated, weighted jacobian
    jacobianw = abs_boundary_jacobian2D[INDEX2(NGLL2,igll,iface)]*fac1;

    // Sommerfeld condition
    atomicAdd(&potential_dot_dot_acoustic[iglob],-sn*jacobianw);

    // adjoint simulations
    if (SAVE_FORWARD){
      // saves boundary values
      b_absorb_potential[INDEX2(NGLL2,igll,iface)] = sn*jacobianw;
    }

  }
}

